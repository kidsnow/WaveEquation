#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "WEgpu.h"
#include "WEmain.h"

const int N = 16;
const int blocksize = 16;

__global__
void hello(char *a, int *b)
{
	a[threadIdx.x] += b[threadIdx.x];
}

int callHelloFromGPU() {
	char a[N] = "Hello \0\0\0\0\0\0";
	int b[N] = { 15, 10, 6, 0, -11, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 };

	char *ad;
	int *bd;
	const int csize = N*sizeof(char);
	const int isize = N*sizeof(int);

	printf("%s", a);

	hipMalloc((void**)&ad, csize);
	hipMalloc((void**)&bd, isize);
	hipMemcpy(ad, a, csize, hipMemcpyHostToDevice);
	hipMemcpy(bd, b, isize, hipMemcpyHostToDevice);

	dim3 dimBlock(blocksize, 1);
	dim3 dimGrid(1, 1);
	hello <<<dimGrid, dimBlock >>>(ad, bd);
	hipMemcpy(a, ad, csize, hipMemcpyDeviceToHost);
	hipFree(ad);
	hipFree(bd);
	 
	printf("%s\n", a);
	return 1;
}

__global__
void computeWave(float diag_el_of_a, float beta, int grid_size, Grid* previous, Grid* current) {
	int x = threadIdx.x + blockIdx.x*blockDim.x;
	int y = threadIdx.y + blockIdx.y*blockDim.y;
	int idx = x + y*blockDim.x*gridDim.x;
	float temp = 0.0;

	if (previous[idx].w == 1.0) {
		temp += previous[idx - grid_size].y;
		temp += previous[idx - 1].y;
		temp += previous[idx + 1].y;
		temp += previous[idx + grid_size].y;
		current[idx].y = (2 * previous[idx].y - current[idx].y + beta*temp) / diag_el_of_a;
	}
}

__global__
void test(float diag_el_of_A, Grid* grid0, Grid* grid1) {
	grid0[0].x += grid1[0].x + diag_el_of_A;
}

void callComputeWave(float diag_el_of_A, float beta, int grid_size, Grid* grid0, Grid* grid1) {
	Grid* GPU_grid0;
	Grid* GPU_grid1;

	hipMalloc(&GPU_grid0, sizeof(Grid)*GRIDSIDENUM*(GRIDSIDENUM + 2));
	hipMalloc(&GPU_grid1, sizeof(Grid)*GRIDSIDENUM*(GRIDSIDENUM + 2));

	hipMemcpy(GPU_grid0, grid0, sizeof(Grid)*GRIDSIDENUM*(GRIDSIDENUM + 2), hipMemcpyHostToDevice);
	hipMemcpy(GPU_grid1, grid1, sizeof(Grid)*GRIDSIDENUM*(GRIDSIDENUM + 2), hipMemcpyHostToDevice);

	computeWave <<<GRIDSIDENUM*(GRIDSIDENUM + 2), GRIDSIDENUM>>> (diag_el_of_A, beta, grid_size, GPU_grid0, GPU_grid1);

	hipMemcpy(grid0, GPU_grid0, sizeof(Grid)*GRIDSIDENUM*(GRIDSIDENUM + 2), hipMemcpyDeviceToHost);
	hipMemcpy(grid1, GPU_grid1, sizeof(Grid)*GRIDSIDENUM*(GRIDSIDENUM + 2), hipMemcpyDeviceToHost);

	return;
}