#include "hip/hip_runtime.h"
#include <stdio.h>
#include "WEgpu.h"
#include "WEmain.h"
#include ""

#define BLOCK_X 32
#define BLOCK_Y 32

__global__
void computeWave(float4 *previous, float4 *current, float4 *previousOfPrevious, float4 *moreEffects, float diag_el_of_A, float beta, int grid_size) {
	const int c = blockIdx.x * blockDim.x + threadIdx.x;
	const int r = blockIdx.y * blockDim.y + threadIdx.y;

	if ((c >= 64) || (r >= 66)) return;
	const int idx = c + r * 64;	// 1D indexing.

	float temp = 0.0;

	if (previous[idx].w == 1.0) {
		temp += previous[idx - grid_size].y;
		temp += previous[idx - 1].y;
		temp += previous[idx + 1].y;
		temp += previous[idx + grid_size].y;
		current[idx].y = (2 * previous[idx].y - previousOfPrevious[idx].y + beta*temp) / diag_el_of_A;
		current[idx].y += moreEffects[idx].y;
		moreEffects[idx].y = 0;
	}
}

int turn2 = 0;

void callComputeWave(float4 **pos_out, float diag_el_of_A, float beta, int grid_size) {
	int deviceCount; hipGetDeviceCount(&deviceCount); int device; for (device = 0; device < deviceCount; ++device) { hipDeviceProp_t deviceProp; hipGetDeviceProperties(&deviceProp, device); printf("Device %d has compute capability %d.%d.\n", device, deviceProp.major, deviceProp.minor); }

	const dim3 blockSize(BLOCK_X, BLOCK_Y);
	const dim3 gridSize = dim3(64 / BLOCK_X, 64 / BLOCK_Y);
	for (int i = 0; i < ITERNUM; i++) {
		computeWave <<<gridSize, blockSize>>>(pos_out[turn2 % 3], pos_out[(turn2+1) % 3], pos_out[(turn2+2) % 3], pos_out[3], diag_el_of_A, beta, grid_size);
		turn2++;
	}
}